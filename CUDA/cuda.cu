#include <stdio.h>
#include <vector>

#pragma comment(lib , "cuda.lib")
#pragma comment(lib, "cudart.lib")
#include <hip/hip_runtime.h>
#include <math.h>

#include <hip/hip_runtime_api.h>

#include <hipblas.h>

#define blocksize 8;

void matrix_read(double *L, int dimension){
    FILE *fp;
    int row, col;

    fp = fopen("matrix1000.txt", "r");

    if(fp == NULL){
        printf("Ayuda puta\n");
        return;
    }
    for(row = 0; row < dimension; row++){
        for(col = 0; col < dimension; col++){
            if(fscanf(fp, "%lf,", &L[row * dimension + col]) == EOF) 
                break;
            
        }
        if(feof(fp)) break;
    }

    fclose(fp);
}

__global__ void normalize(double *A, double *I, int n, int i){
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if(x < n && y < n){
        if(x == i){
            I[x*n + y] /= A[i*n + i];
            A[x*n + y] /= A[i*n + i];
        }
    }
}

__global__ void gaussJordan(double *A, double *I, int n, int i){
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if(x < n && y < n){
        if(x != i){
            I[x*n + y] -= I[i*n + y] * A[x*n + i];
            if(y != i){
                A[x*n + y] -= A[i*n + y] * A[x*n + i];   
            }
        }
    }
}

__global__ void set_zero(double *A, double *I, int n, int i){
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if(x < n && y < n){
        if(x != i){
            if(y == i){
                A[x*n + y] = 0;   
            }
        }
    }
}

void saveToFile(char *filename, double *A, int n, int h){

    FILE *ofile;
    ofile = fopen(filename, "w");
    for(int i = 0; i < h; i++){
        for(int j = 0; j< h; j++){
            fprintf(ofile,"%f\t", A[i*n + j]);
        }
        fprintf(ofile,"\n");
    }
    fclose(ofile);
}


void printMatrix(double *A, int n){
	int x, y;
	for(y = 0; y < n; y++){
		printf("\n");
		for(x = 0; x < n; x++){
		    printf("%f ",  A[y*n + x]);
		}
	}
	printf("\n");
}

int main(){

    const int n = 1000;

    double *iL = new double[n*n];
    double *L = new double[n*n];
    matrix_read(L, n);

    printf("Matriz inversa\n");
    double *d_A, *I, *d_I;
    float time;
    hipError_t err;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    int ddsize = n*n*sizeof(double);

    dim3 threadsPerBlock(8, 8); // blocksize = 8
    dim3 numBlocks(125 , 125); // (n + blocksize -1) / blocksize

    err = hipMalloc((void**)&d_A, ddsize);
    if(err != hipSuccess){ fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err)); }
    err = hipMalloc((void**)&d_I, ddsize);
    if(err != hipSuccess){ fprintf(stderr, "Failed to allocate device vector I (error code %s)!\n", hipGetErrorString(err)); }
    I = new double[n*n];

    for(int i=0; i < n; i++){
        for(int j = 0; j < n; j++){
            if(i == j) I[i*n + j] = 1.0;
            else I[i*n + j] = 0.0;
        }
    }

    err = hipMemcpy(d_A, L, ddsize, hipMemcpyHostToDevice);
    if (err != hipSuccess){ fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err)); }
    err = hipMemcpy(d_I, I, ddsize, hipMemcpyHostToDevice);
    if (err != hipSuccess){ fprintf(stderr, "Failed to copy vector I from host to device (error code %s)!\n", hipGetErrorString(err)); }

    saveToFile("mat.txt" ,L, n, n);

    hipEventRecord(start, 0);


    for(int i= 0; i < n; i++){
        normalize <<<numBlocks, threadsPerBlock >>> (d_A, d_I, n, i);
        gaussJordan <<<numBlocks, threadsPerBlock >>> (d_A, d_I, n, i);
        set_zero <<<numBlocks, threadsPerBlock >>> (d_A, d_I, n, i);
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);


    err = hipMemcpy(iL, d_I, ddsize, hipMemcpyDeviceToHost);
    if (err != hipSuccess){ fprintf(stderr, "Failed to copy vector IL from host to device (error code %s)!\n", hipGetErrorString(err)); }
    err = hipMemcpy(I, d_A, ddsize, hipMemcpyDeviceToHost);
    if (err != hipSuccess){ fprintf(stderr, "Failed to copy vector d_A to I from host to device (error code %s)!\n", hipGetErrorString(err)); }

    printf("Cuda time: %lf ms\n", time);
    saveToFile("inversa_cuda.txt" ,iL, n, n);

    hipFree(d_A);
    hipFree(d_I);

    
    double *c = new double[n*n];
    for(int i= 0 ; i< n; i++){
        for(int j = 0; j < n; j++){
            c[i*n + j] = 0;
            for(int x = 0; x < n; x++){
                c[i*n + j] = c[i*n + j] + L[i*n +x] * iL[x*n + j];
            }
        }
    }
    saveToFile("c.txt" ,c, n, n);

    return 0;
}